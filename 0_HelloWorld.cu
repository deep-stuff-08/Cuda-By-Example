#include "hip/hip_runtime.h"
#include<iostream>
#include"common/book.h"

__global__ void add(int a, int b, int *c) {
	*c = a + b;
}

int main(void) {
	int c;
	int *dev_c;
	HANDLE_ERROR( hipMalloc((void**)&dev_c, sizeof(int)));
	add<<<1, 1>>>(2, 7, dev_c);
	HANDLE_ERROR(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));
	printf("2+7=%d\n", c);
	hipFree(dev_c);
	return 0;
}