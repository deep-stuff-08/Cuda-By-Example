#include "hip/hip_runtime.h"
#include"common/book.h"

#include"common/book.h"

#define iMin(a, b) (a < b)?a:b

const int N = 33 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = iMin(32, (N + threadsPerBlock - 1) / threadsPerBlock);

struct DataStruct {
	int devId;
	int size;
	int offset;
	float *a;
	float *b;
	float retValue;
};

__global__ void dot(int size, float *a, float *b, float *c) {
	__shared__ float cache[threadsPerBlock];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cacheIndex = threadIdx.x;

	float temp = 0;
	while(tid < size) {
		temp += a[tid] * b[tid];
		tid += blockDim.x * gridDim.x;
	}

	cache[cacheIndex] = temp;

	__syncthreads();

	int i = blockDim.x / 2;
	while(i != 0) {
		if(cacheIndex < i) {
			cache[cacheIndex] += cache[cacheIndex + 1];
		}
		__syncthreads();
		i /= 2;
	}

	if(cacheIndex == 0) {
		c[blockIdx.x] = cache[0];
	}
}

void *routine(void *pvoiddata) {
	DataStruct *data = (DataStruct*)pvoiddata;
	if(data->devId != 0) {
		HANDLE_ERROR(hipSetDevice(data->devId));
		HANDLE_ERROR(hipSetDeviceFlags(hipDeviceMapHost));
	}
	
	int size = data->size;
	float *a, *b, c, *partial_c;
	float *dev_a, *dev_b, *dev_partial_c;

	a = data->a;
	b = data->b;
	partial_c = (float*)malloc(sizeof(float) * blocksPerGrid);

	HANDLE_ERROR(hipHostGetDevicePointer(&dev_a, a, 0));
	HANDLE_ERROR(hipHostGetDevicePointer(&dev_b, b, 0));
	HANDLE_ERROR(hipMalloc((void**)&dev_partial_c, sizeof(float) * blocksPerGrid));

	dev_a += data->offset;
	dev_b += data->offset;

	dot<<<blocksPerGrid, threadsPerBlock>>>(size, dev_a, dev_b, dev_partial_c);

	HANDLE_ERROR(hipMemcpy(partial_c, dev_partial_c, sizeof(float) * blocksPerGrid, hipMemcpyDeviceToHost));

	c = 0;
	for(int i = 0; i < blocksPerGrid; i++) {
		c += partial_c[i];
	}

	data->retValue = c;

	hipFree(dev_partial_c);

	free(partial_c);

	return 0;
}

int main(void) {
	float *a, *b;
	
	int devCount;
	HANDLE_ERROR(hipGetDeviceCount(&devCount));
	if(devCount < 2) {
		printf("Not enough CUDA capable devices found.\n");
		return 0;
	}

	HANDLE_ERROR(hipSetDevice(0));
	HANDLE_ERROR(hipSetDeviceFlags(hipDeviceMapHost));
	HANDLE_ERROR(hipHostAlloc((void**) &a, N * sizeof(float), hipHostMallocWriteCombined | hipHostMallocPortable | hipHostMallocMapped));
	HANDLE_ERROR(hipHostAlloc((void**) &b, N * sizeof(float), hipHostMallocWriteCombined | hipHostMallocPortable | hipHostMallocMapped));

	for(int i = 0; i < N; i++) {
		a[i] = i;
		b[i] = i * 2;
	}

	DataStruct data[2];
	data[0].devId = 0;
	data[0].size = N / 2;
	data[0].offset = 0;
	data[0].a = a;
	data[0].b = b;

	data[1].devId = 1;
	data[1].size = N / 2;
	data[1].offset = N / 2;
	data[1].a = a + N/2;
	data[1].b = b + N/2;

	CUTThread thread = start_thread(routine, data);
	routine(data+1);
	end_thread(thread);

	HANDLE_ERROR(hipHostFree(a));
	HANDLE_ERROR(hipHostFree(b));

	printf("Value calculated: %f\n", data[0].retValue + data[1].retValue);
	return 0;
}