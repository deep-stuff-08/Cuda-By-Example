#include "hip/hip_runtime.h"
#include"common/book.h"

#define N (1024 * 1024)
#define DATA_SIZE (N * 20)

__global__ void kernel(int *a, int *b, int *c) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx < N) {
		int idx1 = (idx + 1) % 256;
		int idx2 = (idx + 2) % 256;
		float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
		float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
		c[idx] = (as + bs) / 2;
	}
}

int main(void) {
	hipEvent_t start, stop;
	float elapsedTime;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));

	hipStream_t stream;
	HANDLE_ERROR(hipStreamCreate(&stream));

	int *host_a, *host_b, *host_c;
	int *dev_a, *dev_b, *dev_c;

	HANDLE_ERROR(hipMalloc((void**) &dev_a, sizeof(int) * N));
	HANDLE_ERROR(hipMalloc((void**) &dev_b, sizeof(int) * N));
	HANDLE_ERROR(hipMalloc((void**) &dev_c, sizeof(int) * N));

	HANDLE_ERROR(hipHostAlloc((void**) &host_a, sizeof(int) * DATA_SIZE, hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**) &host_b, sizeof(int) * DATA_SIZE, hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**) &host_c, sizeof(int) * DATA_SIZE, hipHostMallocDefault));

	for(int i = 0; i < DATA_SIZE; i++) {
		host_a[i] = rand();
		host_b[i] = rand();
	}

	HANDLE_ERROR(hipEventRecord(start, 0));
	for(int i = 0; i < DATA_SIZE; i += N) {
		HANDLE_ERROR(hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream));
		HANDLE_ERROR(hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream));
		kernel<<<N/256, 256, 0, stream>>>(dev_a, dev_b, dev_c);
		HANDLE_ERROR(hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, stream));
	}

	HANDLE_ERROR(hipStreamSynchronize(stream));

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

	printf("Time Taken: %3.1f\n", elapsedTime);

	HANDLE_ERROR(hipHostFree(host_a));
	HANDLE_ERROR(hipHostFree(host_b));
	HANDLE_ERROR(hipHostFree(host_c));

	HANDLE_ERROR(hipFree(dev_a));
	HANDLE_ERROR(hipFree(dev_b));
	HANDLE_ERROR(hipFree(dev_c));

	HANDLE_ERROR( hipStreamDestroy( stream ) );
	return 0;
}