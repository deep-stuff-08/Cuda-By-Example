#include "hip/hip_runtime.h"
#include"common/book.h"

#define iMin(a, b) (a < b)?a:b

const int N = 33 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = iMin(32, (N + threadsPerBlock - 1) / threadsPerBlock);

__global__ void dot(float *a, float *b, float *c) {
	__shared__ float cache[threadsPerBlock];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cacheIndex = threadIdx.x;

	float temp = 0;
	while(tid < N) {
		temp += a[tid] * b[tid];
		tid += blockDim.x * gridDim.x;
	}

	cache[cacheIndex] = temp;

	__syncthreads();

	int i = blockDim.x / 2;
	while(i != 0) {
		if(cacheIndex < i) {
			cache[cacheIndex] += cache[cacheIndex + 1];
		}
		__syncthreads();
		i /= 2;
	}

	if(cacheIndex == 0) {
		c[blockIdx.x] = cache[0];
	}
}

int main(void) {
	float *a, *b, *partial_c;
	float *dev_a, *dev_b, *dev_partial_c;
	float c;

	a = (float*)malloc(sizeof(float) * N);
	b = (float*)malloc(sizeof(float) * N);
	partial_c = (float*)malloc(sizeof(float) * blocksPerGrid);

	HANDLE_ERROR(hipMalloc((void**)&dev_a, sizeof(float) * N));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, sizeof(float) * N));
	HANDLE_ERROR(hipMalloc((void**)&dev_partial_c, sizeof(float) * blocksPerGrid));

	for(int i = 0; i < N; i++) {
		a[i] = i;
		b[i] = i * 2;
	}

	HANDLE_ERROR(hipMemcpy(dev_a, a, sizeof(float) * N, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, sizeof(float) * N, hipMemcpyHostToDevice));

	dot<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_partial_c);

	HANDLE_ERROR(hipMemcpy(partial_c, dev_partial_c, sizeof(float) * blocksPerGrid, hipMemcpyDeviceToHost));

	c = 0;
	for(int i = 0; i < blocksPerGrid; i++) {
		c += partial_c[i];
	}

	printf("Dot product = %g\n", c);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_partial_c);

	free(a);
	free(b);
	free(partial_c);
}