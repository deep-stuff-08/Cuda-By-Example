#include "hip/hip_runtime.h"
#include<iostream>
#include"common/book.h"

#define N 65534

__global__ void addVec(int *a, int *b, int *c) {
	int tid = blockIdx.x;
	if(tid < N) {
		c[tid] = a[tid] + b[tid];
	}
}

int main(void) {
	hipEvent_t start, stop;
	
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));

	HANDLE_ERROR(hipEventRecord(start, 0));

	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	HANDLE_ERROR(hipMalloc((void**) &dev_a, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**) &dev_b, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**) &dev_c, N * sizeof(int)));

	for(int i = 1; i <= N; i++) {
		a[i] = i;
		b[i] = i * 2;
	}

	HANDLE_ERROR(hipMemcpy(dev_a, a, sizeof(int) * N, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, sizeof(int) * N, hipMemcpyHostToDevice));

	addVec<<<N, 1>>>(dev_a, dev_b, dev_c);

	HANDLE_ERROR(hipMemcpy(c, dev_c, sizeof(int) * N, hipMemcpyDeviceToHost));
	
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));

	float elapsedTime;
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));
	
	for(int i = 0; i < N; i++) {
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	printf("time required to add two %d-sized vectors: %f\n", N, elapsedTime);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}