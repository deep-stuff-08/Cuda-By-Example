#include "hip/hip_runtime.h"
#include<iostream>
#include<stdlib.h>
#include<unistd.h>
#include"common/book.h"


#define N 300000000

__global__ void reallyLongVecAdd(int *a, int *b, int *c) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	while(tid < N) {
		c[tid] = a[tid] + b[tid];
		tid += blockDim.x * gridDim.x;
	}
}

int main(void) {
	int *a, *b, *c;
	int *dev_a, *dev_b, *dev_c;

	a = (int*)malloc(sizeof(int) * N);
	b = (int*)malloc(sizeof(int) * N);
	c = (int*)malloc(sizeof(int) * N);

	HANDLE_ERROR(hipMalloc((void**) &dev_a, sizeof(int) * N));
	HANDLE_ERROR(hipMalloc((void**) &dev_b, sizeof(int) * N));
	HANDLE_ERROR(hipMalloc((void**) &dev_c, sizeof(int) * N));

	for(int i = 0; i < N; i++) {
		a[i] = i;
		b[i] = N - i;
	}

	HANDLE_ERROR(hipMemcpy(dev_a, a, sizeof(int) * N, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, sizeof(int) * N, hipMemcpyHostToDevice));

	reallyLongVecAdd<<<128, 128>>>(dev_a, dev_b, dev_c);

	HANDLE_ERROR(hipMemcpy(c, dev_c, sizeof(int) * N, hipMemcpyDeviceToHost));

	for(int i = 0; i < N; i += 100) {
		printf("No. %d -> %d + %d = %d\n", i, a[i], b[i], c[i]);
	}
	sleep(100000000);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}