#include "hip/hip_runtime.h"
#include"common/book.h"

#define N (1024 * 1024)
#define DATA_SIZE (N * 20)

__global__ void kernel(int *a, int *b, int *c) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx < N) {
		int idx1 = (idx + 1) % 256;
		int idx2 = (idx + 2) % 256;
		float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
		float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
		c[idx] = (as + bs) / 2;
	}
}

int main(void) {
	hipEvent_t start, stop;
	float elapsedTime;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));

	hipStream_t stream0, stream1;
	HANDLE_ERROR(hipStreamCreate(&stream0));
	HANDLE_ERROR(hipStreamCreate(&stream1));

	int *host_a, *host_b, *host_c;
	int *dev_a0, *dev_b0, *dev_c0;
	int *dev_a1, *dev_b1, *dev_c1;

	HANDLE_ERROR(hipMalloc((void**) &dev_a0, sizeof(int) * N));
	HANDLE_ERROR(hipMalloc((void**) &dev_b0, sizeof(int) * N));
	HANDLE_ERROR(hipMalloc((void**) &dev_c0, sizeof(int) * N));
	HANDLE_ERROR(hipMalloc((void**) &dev_a1, sizeof(int) * N));
	HANDLE_ERROR(hipMalloc((void**) &dev_b1, sizeof(int) * N));
	HANDLE_ERROR(hipMalloc((void**) &dev_c1, sizeof(int) * N));

	HANDLE_ERROR(hipHostAlloc((void**) &host_a, sizeof(int) * DATA_SIZE, hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**) &host_b, sizeof(int) * DATA_SIZE, hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**) &host_c, sizeof(int) * DATA_SIZE, hipHostMallocDefault));

	for(int i = 0; i < DATA_SIZE; i++) {
		host_a[i] = rand();
		host_b[i] = rand();
	}

	HANDLE_ERROR(hipEventRecord(start, 0));
	for(int i = 0; i < DATA_SIZE; i += N * 2) {
		HANDLE_ERROR(hipMemcpyAsync(dev_a0, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream0));
		HANDLE_ERROR(hipMemcpyAsync(dev_a1, host_a + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1));

		HANDLE_ERROR(hipMemcpyAsync(dev_b0, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream0));
		HANDLE_ERROR(hipMemcpyAsync(dev_b1, host_b + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1));

		kernel<<<N/256, 256, 0, stream0>>>(dev_a0, dev_b0, dev_c0);
		kernel<<<N/256, 256, 0, stream1>>>(dev_a1, dev_b1, dev_c1);

		HANDLE_ERROR(hipMemcpyAsync(host_c + i, dev_c0, N * sizeof(int), hipMemcpyDeviceToHost, stream0));	
		HANDLE_ERROR(hipMemcpyAsync(host_c + i  + N, dev_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream1));
	}

	HANDLE_ERROR(hipStreamSynchronize(stream0));
	HANDLE_ERROR(hipStreamSynchronize(stream1));

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

	printf("Time Taken: %3.1f\n", elapsedTime);

	HANDLE_ERROR(hipHostFree(host_a));
	HANDLE_ERROR(hipHostFree(host_b));
	HANDLE_ERROR(hipHostFree(host_c));

	HANDLE_ERROR(hipFree(dev_a0));
	HANDLE_ERROR(hipFree(dev_b0));
	HANDLE_ERROR(hipFree(dev_c0));
	HANDLE_ERROR(hipFree(dev_a1));
	HANDLE_ERROR(hipFree(dev_b1));
	HANDLE_ERROR(hipFree(dev_c1));

	HANDLE_ERROR(hipStreamDestroy(stream0));
	HANDLE_ERROR(hipStreamDestroy(stream1));
	return 0;
}