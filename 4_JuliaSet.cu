#include "hip/hip_runtime.h"
#include"common/book.h"
#include"common/cpu_anim.h"

#define DIM 1000

struct hipComplex {
	float r;
	float i;

	__device__ hipComplex(float a, float b): r(a), i(b) { 
	}

	__device__ float magnitude2() {
		return r * r + i * i;
	}

	__device__ hipComplex operator*(hipComplex& c) {
		return hipComplex(r * c.r - i * c.i, i * c.r + r * c.i);
	}

	__device__ hipComplex operator+(hipComplex& c) {
		return hipComplex(r + c.r, i + c.i);
	}
};

__device__ int julia(int x, int y, float time) {
	const float scale = 1.5f;
	float jx = scale * (float)(DIM / 2 - x) / (DIM / 2);
	float jy = scale * (float)(DIM / 2 - y) / (DIM / 2);

	hipComplex c(time, time);
	hipComplex a(jx, jy);

	int i = 0;
	for(i = 0; i < 200; i++) {
		a = a * a + c;
		if(a.magnitude2() > 1000) {
			return 1;
		}
	}
	return 0;
}

__global__ void kernel(unsigned char *ptr, float time) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offset = x + y * gridDim.x;
	
	int isJulia = julia(x, y, time);

	ptr[offset*4 + 0] = 255 * isJulia;
	ptr[offset*4 + 1] = 0;
	ptr[offset*4 + 2] = 0;
	ptr[offset*4 + 3] = 255;
}

struct DataBlock {
	unsigned char *dev_bitmap;
	CPUAnimBitmap *bitmap;
};

void cleanup(DataBlock *d) {
	hipFree(d->dev_bitmap);
}

void generateFrame(DataBlock *d, int ticks) {
	static float time = 0.0f;
	static int sign = 1;
	dim3 grid(DIM, DIM);
	kernel<<<grid, 1>>>(d->dev_bitmap, time);
	time += 0.001f * sign;
	if(time >= 0.4f || time <= 0.0f) {
		sign *= -1;
	}
	
	HANDLE_ERROR(hipMemcpy(d->bitmap->get_ptr(), d->dev_bitmap, d->bitmap->image_size(), hipMemcpyDeviceToHost));
}

int main(void) {
	DataBlock data;
	CPUAnimBitmap bitmap(DIM, DIM, &data);
	data.bitmap = &bitmap;

	HANDLE_ERROR(hipMalloc((void**) &data.dev_bitmap, bitmap.image_size()));
	bitmap.anim_and_exit((void (*)(void*, int))generateFrame, (void (*) (void*))cleanup);
}
