#include "hip/hip_runtime.h"
#include"common/book.h"
#include"common/cpu_bitmap.h"
#include"common/gpu_anim.h"

#define DIM 1024

GLuint bufferObj;
hipGraphicsResource *resource;

void __global__ kernel(uchar4 *ptr) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	float fx = x/(float)DIM - 0.5f;
	float fy = y/(float)DIM - 0.5f;
	unsigned char green = 128 + 127 * sin(abs(fx * 100) - abs(fy * 100));

	ptr[offset].x = 0;
	ptr[offset].y = green;
	ptr[offset].z = 0;
	ptr[offset].w = 255; 
}

void draw_func(void) {
	glDrawPixels(DIM, DIM, GL_RGBA, GL_UNSIGNED_BYTE, 0);
	glutSwapBuffers();
}

void key_func(unsigned char key, int x, int y) {
	switch(key) {
	case 27:
		HANDLE_ERROR(hipGraphicsUnregisterResource(resource));
		glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);
		glDeleteBuffers(1, &bufferObj);
		exit(0);
	}
}

int main(int argc, char** argv) {
	hipDeviceProp_t prop;
	int dev;

	memset(&prop, 0, sizeof(hipDeviceProp_t));
	prop.major = 1;
	prop.minor = 0;
	HANDLE_ERROR(hipChooseDevice(&dev, &prop));
	HANDLE_ERROR(cudaGLSetGLDevice(dev));

	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
	glutInitWindowSize(DIM, DIM);
	glutCreateWindow("bitmap");

	glGenBuffers(1, &bufferObj);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, bufferObj);
	glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, DIM * DIM * 4, NULL, GL_DYNAMIC_DRAW_ARB);

	HANDLE_ERROR(hipGraphicsGLRegisterBuffer(&resource, bufferObj, cudaGraphicsMapFlagsNone));

	uchar4* devPtr;
	size_t size;
	HANDLE_ERROR(hipGraphicsMapResources(1, &resource, NULL));
	HANDLE_ERROR(hipGraphicsResourceGetMappedPointer((void**)&devPtr, &size, resource));

	dim3 grids(DIM/16, DIM/16);
	dim3 threads(16, 16);
	kernel<<<grids, threads>>>(devPtr);
	HANDLE_ERROR(hipGraphicsUnmapResources(1, &resource, NULL));

	glutKeyboardFunc(key_func);
	glutDisplayFunc(draw_func);
	glutMainLoop();
}
