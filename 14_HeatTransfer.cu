#include "hip/hip_runtime.h"
#include"common/book.h"
#include"common/cpu_anim.h"

#define DIM 1024
#define PI 3.1415926535897932
#define MAX_TEMP 1.0f
#define MIN_TEMP 0.0001f
#define SPEED 0.25f

struct DataBlock {
	unsigned char* op_bitmap;
	float* dev_InSrc;
	float* dev_OutSrc;
	float* dev_ConstSrc;
	CPUAnimBitmap *bitmap;
	hipEvent_t start, stop;
	float totalTimes;
	float frames;
};

__global__ void copy_const_kernel(float* iPtr, const float* cPtr) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	if(cPtr[offset] != 0) {
		iPtr[offset] = cPtr[offset];
	}
}

__global__ void blend_kernel(float *outSrc, const float *inSrc) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	int left = offset - 1;
	int right = offset + 1;
	if(x == 0) {
		left++;
	}
	if(x == DIM - 1) {
		right--;
	}

	int top = offset - DIM;
	int bottom = offset + DIM;
	if(y == 0) {
		top += DIM;
	}
	if(y == DIM - 1) {
		bottom -= DIM;
	}

	outSrc[offset] = inSrc[offset] + SPEED * (inSrc[top] + inSrc[bottom] + inSrc[left] + inSrc[right] - inSrc[offset] * 4);
}

void anim_gpu(DataBlock *d, int ticks) {
	HANDLE_ERROR(hipEventRecord(d->start, 0));
	dim3 blocks(DIM/16, DIM/16);
	dim3 threads(16, 16);
	CPUAnimBitmap *bitmap = d->bitmap;

	for(int i = 0; i < 90; i++) {
		copy_const_kernel<<<blocks, threads>>>(d->dev_InSrc, d->dev_ConstSrc);
		blend_kernel<<<blocks, threads>>>(d->dev_OutSrc, d->dev_InSrc);
		swap(d->dev_InSrc, d->dev_OutSrc);
	}
	float_to_color<<<blocks, threads>>>(d->op_bitmap, d->dev_InSrc);
	HANDLE_ERROR(hipMemcpy(bitmap->get_ptr(), d->op_bitmap, bitmap->image_size(), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipEventRecord(d->stop, 0));
	HANDLE_ERROR(hipEventSynchronize(d->stop));
	float elapsedTime;
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, d->start, d->stop));
	d->totalTimes += elapsedTime;
	++d->frames;
	printf("Average Time per Frame: %3.1f ms\n", d->totalTimes / d->frames);
}

void anim_exit(DataBlock *d) {
	hipFree(d->dev_InSrc);
	hipFree(d->dev_OutSrc);
	hipFree(d->dev_ConstSrc);

	HANDLE_ERROR(hipEventDestroy(d->start));
	HANDLE_ERROR(hipEventDestroy(d->stop));
}

int main(void) {
	DataBlock data;
	CPUAnimBitmap bitmap(DIM, DIM, &data);
	data.bitmap = &bitmap;
	data.totalTimes = 0;
	data.frames = 0;
	HANDLE_ERROR(hipEventCreate(&data.start));
	HANDLE_ERROR(hipEventCreate(&data.stop));

	HANDLE_ERROR(hipMalloc((void**) &data.op_bitmap, bitmap.image_size()));
	HANDLE_ERROR(hipMalloc((void**) &data.dev_InSrc, bitmap.image_size()));
	HANDLE_ERROR(hipMalloc((void**) &data.dev_OutSrc, bitmap.image_size()));
	HANDLE_ERROR(hipMalloc((void**) &data.dev_ConstSrc, bitmap.image_size()));

	float *temp = (float*)malloc(bitmap.image_size());
	for(int i = 0; i < DIM * DIM; i++) {
		temp[i] = 0;
		int x = i % DIM;
		int y = i / DIM;
		if((x > 300) && (x < 600) && (y > 310) && (y < 601)) {
			temp[i] = MAX_TEMP;
		}
	}
	temp[DIM * 100 + 100] = (MAX_TEMP + MIN_TEMP) / 2;
	temp[DIM * 700 + 100] = MIN_TEMP;
	temp[DIM * 300 + 300] = MIN_TEMP;
	temp[DIM * 200 + 700] = MIN_TEMP;
	for(int y = 800; y < 900; y++) {
		for(int x = 400; x < 500; x++) {
			temp[x + y * DIM] = MIN_TEMP;
		}
	}
	HANDLE_ERROR(hipMemcpy(data.dev_ConstSrc, temp, bitmap.image_size(), hipMemcpyHostToDevice));
	free(temp);

	bitmap.anim_and_exit((void (*)(void*, int))anim_gpu, (void (*)(void*))anim_exit);
}