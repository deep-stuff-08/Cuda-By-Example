#include "hip/hip_runtime.h"
#include<iostream>
#include"common/book.h"

#define N 512

__global__ void vecAdd(int *a, int *b, int *c) {
	int tid = threadIdx.x;
	if(tid < N) {
		c[tid] = a[tid] + b[tid];
	}
}

int main(void) {
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	HANDLE_ERROR(hipMalloc((void**) &dev_a, sizeof(int) * N));
	HANDLE_ERROR(hipMalloc((void**) &dev_b, sizeof(int) * N));
	HANDLE_ERROR(hipMalloc((void**) &dev_c, sizeof(int) * N));

	for(int i = 0; i < N; i++) {
		a[i] = i;
		b[i] = i * 2;
	}

	HANDLE_ERROR(hipMemcpy(dev_a, a, sizeof(int) * N, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, sizeof(int) * N, hipMemcpyHostToDevice));

	vecAdd<<<1, N>>>(dev_a, dev_b, dev_c);

	HANDLE_ERROR(hipMemcpy(c, dev_c, sizeof(int) * N, hipMemcpyDeviceToHost));

	for(int i = 0; i < N; i++) {
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	HANDLE_ERROR(hipFree(dev_a));
	HANDLE_ERROR(hipFree(dev_b));
	HANDLE_ERROR(hipFree(dev_c));
}
