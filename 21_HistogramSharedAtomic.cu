#include "hip/hip_runtime.h"
#include"common/book.h"

#define SIZE (100 * 1024 * 1024)

__global__ void histo_kernel(unsigned char* buffer, long size, unsigned int* histo) {
	__shared__ unsigned int temp[256];
	temp[threadIdx.x] = 0;
	__syncthreads();

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	while(i < size) {
		atomicAdd(&temp[buffer[i]], 1);
		i += stride;
	}

	__syncthreads();
	atomicAdd(&histo[buffer[threadIdx.x]], temp[threadIdx.x]);
}

int main(void) {
	unsigned char* buffer = (unsigned char*)big_random_block(SIZE);
	unsigned int histo[256];
	memset(histo, 0, 256);

	hipEvent_t start, stop;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, 0));

	unsigned char* dev_buffer;
	unsigned int* dev_histo;
	HANDLE_ERROR(hipMalloc((void**) &dev_buffer, SIZE));
	HANDLE_ERROR(hipMemcpy(dev_buffer, buffer, SIZE, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMalloc((void**) &dev_histo, sizeof(int) * 256));
	HANDLE_ERROR(hipMemset(dev_histo, 0, sizeof(int) * 256));

	hipDeviceProp_t prop;
	HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));
	int blocks = prop.multiProcessorCount * 2;
	histo_kernel<<<blocks, 256>>>(dev_buffer, SIZE, dev_histo);

	HANDLE_ERROR(hipMemcpy(histo, dev_histo, sizeof(int) * 256, hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	float time;
	HANDLE_ERROR(hipEventElapsedTime(&time, start, stop));
	printf("Time required = %3.1f ms\n", time);

	long histoCount = 0;
	for(int i = 0; i < 256; i++) {
		histoCount += histo[i];
	}
	if(histoCount != SIZE) {
		printf("Histogram Failed\n");
		for(int i = 0; i < SIZE; i++) {
			histo[buffer[i]]--;
		}
		for(int i = 0; i < 256; i++) {
			if(histo[i] != 0) {
				printf("Failure At %d\n", i+1);
			}
		}
	}
	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));
	HANDLE_ERROR(hipFree(dev_buffer));
	HANDLE_ERROR(hipFree(dev_histo));
	free(buffer);
}