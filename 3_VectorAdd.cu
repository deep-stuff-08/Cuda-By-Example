#include "hip/hip_runtime.h"
#include<iostream>
#include"common/book.h"

#define N 65534

__global__ void addVec(int *a, int *b, int *c) {
	int tid = blockIdx.x;
	if(tid < N) {
		c[tid] = a[tid] + b[tid];
	}
}

int main(void) {
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	HANDLE_ERROR(hipMalloc((void**) &dev_a, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**) &dev_b, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**) &dev_c, N * sizeof(int)));

	for(int i = 1; i <= N; i++) {
		a[i] = i;
		b[i] = i * 2;
	}

	HANDLE_ERROR(hipMemcpy(dev_a, a, sizeof(int) * N, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, sizeof(int) * N, hipMemcpyHostToDevice));

	addVec<<<N, 1>>>(dev_a, dev_b, dev_c);

	HANDLE_ERROR(hipMemcpy(c, dev_c, sizeof(int) * N, hipMemcpyDeviceToHost));
	for(int i = 0; i < N; i++) {
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}