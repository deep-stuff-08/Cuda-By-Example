#include "hip/hip_runtime.h"
#include"common/book.h"
#include"common/cpu_anim.h"

#define DIM 1024
#define PI 3.1415926535897932
#define MAX_TEMP 1.0f
#define MIN_TEMP 0.0001f
#define SPEED 0.25001f

struct DataBlock {
	unsigned char* op_bitmap;
	float* dev_InSrc;
	float* dev_OutSrc;
	float* dev_ConstSrc;
	CPUAnimBitmap *bitmap;
	hipEvent_t start, stop;
	float totalTimes;
	float frames;
};

texture<float, 2> texIn;
texture<float, 2> texOut;
texture<float, 2> texConst;

__global__ void copy_const_kernel(float* iPtr) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	float c = tex2D(texConst, x, y);
	if(c != 0) {
		iPtr[offset] = c;
	}
}

__global__ void blend_kernel(float *dst, bool dstOut) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	float t, l, c, r, b;
	if(dstOut) {
		t = tex2D(texIn, x, y-1);
		l = tex2D(texIn, x-1, y);
		c = tex2D(texIn, x, y);
		r = tex2D(texIn, x+1, y);
		b = tex2D(texIn, x, y+1);
	} else {
		t = tex2D(texOut, x, y-1);
		l = tex2D(texOut, x-1, y);
		c = tex2D(texOut, x, y);
		r = tex2D(texOut, x+1, y);
		b = tex2D(texOut, x, y+1);
	}
	dst[offset] = c + SPEED * (t + b + l + r - c * 4);
}

void anim_gpu(DataBlock *d, int ticks) {
	HANDLE_ERROR(hipEventRecord(d->start, 0));
	dim3 blocks(DIM/16, DIM/16);
	dim3 threads(16, 16);
	CPUAnimBitmap *bitmap = d->bitmap;

	volatile bool dstOut = true;
	for(int i = 0; i < 90; i++) {
		float* in,* out;
		if(dstOut) {
			in = d->dev_InSrc;
			out = d->dev_OutSrc;
		} else {
			in = d->dev_OutSrc;
			out = d->dev_InSrc;
		}
		copy_const_kernel<<<blocks, threads>>>(in);
		blend_kernel<<<blocks, threads>>>(out, dstOut);
		dstOut = ! dstOut;
	}
	float_to_color<<<blocks, threads>>>(d->op_bitmap, d->dev_InSrc);
	HANDLE_ERROR(hipMemcpy(bitmap->get_ptr(), d->op_bitmap, bitmap->image_size(), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipEventRecord(d->stop, 0));
	HANDLE_ERROR(hipEventSynchronize(d->stop));
	float elapsedTime;
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, d->start, d->stop));
	d->totalTimes += elapsedTime;
	++d->frames;
	printf("Average Time per Frame: %3.1f ms\n", d->totalTimes / d->frames);
}

void anim_exit(DataBlock *d) {
	hipUnbindTexture(texIn);
	hipUnbindTexture(texOut);
	hipUnbindTexture(texConst);

	hipFree(d->dev_InSrc);
	hipFree(d->dev_OutSrc);
	hipFree(d->dev_ConstSrc);

	HANDLE_ERROR(hipEventDestroy(d->start));
	HANDLE_ERROR(hipEventDestroy(d->stop));
}

int main(void) {
	DataBlock data;
	CPUAnimBitmap bitmap(DIM, DIM, &data);
	data.bitmap = &bitmap;
	data.totalTimes = 0;
	data.frames = 0;
	HANDLE_ERROR(hipEventCreate(&data.start));
	HANDLE_ERROR(hipEventCreate(&data.stop));

	HANDLE_ERROR(hipMalloc((void**) &data.op_bitmap, bitmap.image_size()));
	HANDLE_ERROR(hipMalloc((void**) &data.dev_InSrc, bitmap.image_size()));
	HANDLE_ERROR(hipMalloc((void**) &data.dev_OutSrc, bitmap.image_size()));
	HANDLE_ERROR(hipMalloc((void**) &data.dev_ConstSrc, bitmap.image_size()));

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	HANDLE_ERROR(hipBindTexture2D(NULL, texConst, data.dev_ConstSrc, desc, DIM, DIM, sizeof(float) * DIM));
	HANDLE_ERROR(hipBindTexture2D(NULL, texIn, data.dev_InSrc, desc, DIM, DIM, sizeof(float) * DIM));
	HANDLE_ERROR(hipBindTexture2D(NULL, texOut, data.dev_OutSrc, desc, DIM, DIM, sizeof(float) * DIM));

	float *temp = (float*)malloc(bitmap.image_size());
	for(int i = 0; i < DIM * DIM; i++) {
		temp[i] = 0;
		int x = i % DIM;
		int y = i / DIM;
		if((x > 300) && (x < 600) && (y > 310) && (y < 601)) {
			temp[i] = MAX_TEMP;
		}
	}
	temp[DIM * 100 + 100] = (MAX_TEMP + MIN_TEMP) / 2;
	temp[DIM * 700 + 100] = MIN_TEMP;
	temp[DIM * 300 + 300] = MIN_TEMP;
	temp[DIM * 200 + 700] = MIN_TEMP;
	for(int y = 800; y < 900; y++) {
		for(int x = 400; x < 500; x++) {
			temp[x + y * DIM] = MIN_TEMP;
		}
	}
	HANDLE_ERROR(hipMemcpy(data.dev_ConstSrc, temp, bitmap.image_size(), hipMemcpyHostToDevice));
	free(temp);

	bitmap.anim_and_exit((void (*)(void*, int))anim_gpu, (void (*)(void*))anim_exit);
}