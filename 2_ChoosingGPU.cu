#include<iostream>
#include"common/book.h"

int main(void) {
	hipDeviceProp_t prop;
	int dev;
	HANDLE_ERROR(hipGetDevice(&dev));
	printf("Current Device: %d\n", dev);
	memset(&prop, 0, sizeof(hipDeviceProp_t));
	prop.major = 6;
	prop.minor = 5;
	HANDLE_ERROR(hipChooseDevice(&dev, &prop));
	printf("Closest Device: %d\n", dev);
	HANDLE_ERROR(hipSetDevice(dev));
}
